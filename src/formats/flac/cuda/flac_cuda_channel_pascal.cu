#include "hip/hip_runtime.h"
/*
 * This file is part of GPUraku.
 * 
 * GPUraku is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 2 of the License, or
 * (at your option) any later version.
 * 
 * GPUraku is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with GPUraku.  If not, see <http://www.gnu.org/licenses/>.
 */

// This is the channel assignment codes for GTX1080

__global__ void flac_cuda_left_assignment(
    grint32 *leftChannel, 
    grint32 *rightChannel)
{
    //The right channel is not the original data.
    rightChannel[threadIdx.x]=leftChannel[threadIdx.x]-rightChannel[threadIdx.x];
}

__global__ void flac_cuda_right_assignment(
    grint32 *leftChannel, 
    grint32 *rightChannel)
{
    //The right channel is not the original data.
    leftChannel[threadIdx.x]+=rightChannel[threadIdx.x];
}

__global__ void flac_cuda_mid_assignment(
    grint32 *leftChannel, 
    grint32 *rightChannel)
{
    //The right channel is not the original data.
    grint32 side=rightChannel[threadIdx.x], 
            mid=(leftChannel[threadIdx.x]<<1) | (side & 1);
    leftChannel[threadIdx.x]=(mid+side)>>1;
    rightChannel[threadIdx.x]=(mid-side)>>1;
}

__global__ void flac_cuda_decorrelate_interchannel(
    CudaFrameDecode *decodeData,
    gruint32 frameCount)
{
    //Calculate the core index.
    int threadId=blockDim.x * blockIdx.x + threadIdx.x;
    if(threadId >= frameCount)
    {
        return;
    }
    //Get the sub frame type.
    gruint8 channelAssignment=decodeData[threadId].channelAssignment;
    if(channelAssignment==FLAC_CHANNEL_INDEPENDENT)
    {
        //No need to do any thing.
        return;
    }
    //Quick calculate the block size.
    gruint32 blockSize=decodeData[threadId].blockSize;
    //Get the channel pointer.
    grint32 *left=decodeData[threadId].channelPcm[0],
            *right=decodeData[threadId].channelPcm[1];
    if(channelAssignment==FLAC_CHANNEL_LEFT_ASSIGNMENT)
    {
        flac_cuda_left_assignment<<<1, blockSize>>>(left, right);
        return;
    }
    if(channelAssignment==FLAC_CHANNEL_MID_ASSIGNMENT)
    {
        flac_cuda_mid_assignment<<<1, blockSize>>>(left, right);
        return;
    }
    //FLAC_CHANNEL_RIGHT_ASSIGNMENT
    flac_cuda_right_assignment<<<1, blockSize>>>(left, right);
}